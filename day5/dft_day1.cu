#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <stdio.h>

// Ajout de extern "C" pour l'interfaçage avec Python
extern "C" {
    void compute_dft(hipDoubleComplex* h_input, hipDoubleComplex* h_output, int N);
}

__global__ void dft_kernel(hipDoubleComplex* x, hipDoubleComplex* X, int N) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;

    if (k < N) {
        hipDoubleComplex sum = make_hipDoubleComplex(0.0, 0.0);

        for (int n = 0; n < N; n++) {
            double angle = -2.0 * M_PI * k * n / N;
            hipDoubleComplex w = make_hipDoubleComplex(cos(angle), sin(angle));
            sum = hipCadd(sum, hipCmul(x[n], w));
        }

        X[k] = sum;
    }
}

void compute_dft(hipDoubleComplex* h_input, hipDoubleComplex* h_output, int N) {
    hipDoubleComplex *d_input, *d_output;

    hipMalloc(&d_input, N * sizeof(hipDoubleComplex));
    hipMalloc(&d_output, N * sizeof(hipDoubleComplex));

    hipMemcpy(d_input, h_input, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    dft_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, N);

    hipMemcpy(h_output, d_output, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}