
#include <hip/hip_runtime.h>
#include <stdio.h>

// __global__ signifie qu'il s'agit d'un kernel CUDA
__global__ void afficherThreadID() {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    printf("Thread ID : %d\n", id);
}

int main() {
    printf("Hello World !\n");
    int nbBlocs = 2;
    int threadsParBloc = 5;

    afficherThreadID<<<nbBlocs, threadsParBloc>>>();
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Erreur de lancement: %s\n", hipGetErrorString(err));
        return -1;
    }
    
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("Erreur d'exécution: %s\n", hipGetErrorString(err));
        return -1;
    }

    return 0;
}
